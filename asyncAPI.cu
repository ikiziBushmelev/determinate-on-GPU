#include "hip/hip_runtime.h"
#ifndef CUDACC
#define CUDACC
#endif
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <conio.h>


//blockDim - ������ �����
//blockIdx - ������ �������� �����
//threadIdx - ������ ������� ���� � �����


__global__ void test(double* arr, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    double kof;
    for (j = 0; j < N; j++)
    {
        if (i >= j && i < N - 1)
        {
            kof = arr[(i + 1) * N + j] / arr[j * N + j];
            int g = blockIdx.y * blockDim.y + threadIdx.y;
            //printf("g = %d i = %d j = %d  kof = %.6f\n", g, i, j, kof);
            //__syncthreads();

            if (g < N)
            {
                arr[(i + 1) * N + g] -= kof * arr[j * N + g];
                //printf("g posle = %d\n", g);
                printf(".");
            }

        }
        // __syncthreads();
    }
    // arr[0] = 10;
}

__host__ int main()
{
    int N;
    printf("Enrer size of matrix N = ");
    scanf_s("%i", &N);
    int SizeMatr = N * N;//������ �������
    int SizeInByte = SizeMatr * sizeof(double);//������, ����������� ��� ������� �� GPU 
    double* pMatr = new double[SizeMatr];//�������� ������ ��� ������

    //��������� ������� ���������� ������� � ������� 
    srand(time(NULL));
    for (int i = 0; i < SizeMatr; i++)
    {
        pMatr[i] = 1 + rand() % 9;
    }
    pMatr[0] = 8;
    pMatr[1] = 6;
    pMatr[2] = 6;
    pMatr[3] = 8;
    pMatr[4] = 1;
    pMatr[5] = 9;
    pMatr[6] = 8;
    pMatr[7] = 5;
    pMatr[8] = 3;
    printf("\n");
    for (int i = 0; i < SizeMatr; i++)
    {
        printf("%0.2f ", pMatr[i]);
        if (((i + 1) % N == 0) && (i != 0)) printf("\n");
    }
    printf("\n");


    double* pMatr_GPU;

    hipMalloc((void**)&pMatr_GPU, SizeInByte);//�������� ������ ��� ������ �� GPU
    hipMemcpy(pMatr_GPU, pMatr, SizeInByte, hipMemcpyHostToDevice);//�������� �������� ������� �� GPU 

    dim3 gridSize = dim3(N, N, 1);//����������� ����� ������ (dim3), ���������� ��� ��������
    dim3 blockSize = dim3(1, 1, 1);//������ ����� (dim3), ����������� ��� ��������


    test << < gridSize, blockSize >> > (pMatr_GPU, N); // ����� ������� ��� ��������� ������� 

    hipDeviceSynchronize();//�������������� ������
    hipMemcpy(pMatr, pMatr_GPU, SizeInByte, hipMemcpyDeviceToHost);//�������� ����� ������� � GPU ������� �� CPU
    printf("\n");
    for (int i = 0; i < SizeMatr; i++)  //������� ���������� �������
    {
        printf("%0.2f ", pMatr[i]);
        if (((i + 1) % N == 0) && (i != 0)) printf("\n");
    }
    printf("\n");

    return 0;
}
