#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>

__global__ void test(double* arr, int N)
{
    double x = 10;
    arr[N] = x+5;
    arr[0] = x;
}

__host__ int main()
{
    int N;
    printf("Enrer size of matrix N = ");
    scanf_s("%i", &N);
    int SizeMatr = N * N;//������ �������
    int SizeInByte = SizeMatr * sizeof(double);//������, ����������� ��� ������� �� GPU 
    double* pMatr = new double[SizeMatr];//�������� ������ ��� ������

    //��������� ������� ���������� ������� � ������� 
    srand(time(NULL));
    for (int i = 0; i < SizeMatr; i++)
    {
        pMatr[i] = 1 + rand() % 9;
    }
    printf("\n");
    for (int i = 0; i < SizeMatr; i++)
    {
        printf("%0.2f ", pMatr[i]);
        if (((i + 1) % N == 0) && (i != 0)) printf("\n");
    }
    printf("\n");


    double* pMatr_GPU;

    hipMalloc((void**)&pMatr_GPU, SizeInByte);//�������� ������ ��� ������ �� GPU
    hipMemcpy(pMatr_GPU, pMatr, SizeInByte, hipMemcpyHostToDevice);//�������� �������� ������� �� GPU 

    dim3 gridSize = dim3(N, N, 1);//����������� ����� ������ (dim3), ���������� ��� ��������
    dim3 blockSize = dim3(1, 1, 1);//������ ����� (dim3), ����������� ��� ��������

    
    test << < gridSize, blockSize >> > (pMatr_GPU, N); // �������� ����� ������� ��� ��������� ������� 

    hipDeviceSynchronize();//�������������� ������
    hipMemcpy(pMatr, pMatr_GPU, SizeInByte, hipMemcpyDeviceToHost);//�������� ����� ������� � GPU ������� �� CPU

    for (int i = 0; i < SizeMatr; i++)  //������� ���������� �������
    {
        printf("%0.2f ", pMatr[i]); 
        if (((i + 1) % N == 0) && (i != 0)) printf("\n");
    }
    printf("\n");
   
    return 0;
}
